#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "../util/CUDAEventTimer.h"
#include "../util/Timer.h"
#include "../util/compute_blocks.h"

typedef float REAL_T;

surface<void, 3> in_surface;
surface<void, 3> out_surface;

#include "../util/do_all_3d.h"
#include "../util/compute.h"
#include "../util/stencils.h"
#include "../util/cuda_error.h"   

template <typename T>
struct distance {
    distance(T eps = T()) : epsilon(eps) {}
    bool operator()(const T& v1, const T& v2) const {
        return std::abs(v1 - v2) <= epsilon;
    }
    T epsilon;
};

std::ostream& operator<<(std::ostream& os, const dim3 d) {
    os << d.x << ' ' << d.y << ' ' << d.z;
    return os;
}

#define cc { \
    std::cout << __LINE__ << std::endl; \
}

REAL_T EPS = REAL_T(0.000001);

int main(int argc, char** argv) {
    hipDeviceReset();
    if(argc < 8) {
        std::cout << "usage: " << argv[0]
                  << " width height depth <threads per block x y z> nsteps "
                     "[iteration axis]" << std::endl;
        return 1;
    }
    char axis = 0;
    //if axis is set then the launch configuration is done on a 2d slice
    //and the kernel performs the iteration over the specified axis
    if(argc == 9 ) {
        axis = argv[8][0];
        if(axis != 'x' && axis != 'y' && axis != 'z') {
            std::cout << "axis must be one of either 'x', 'y', 'z'"
                      << std::endl;
            return 1;
        }
    } 

    //temporary for ease of testing with COSMO
    const int ioffset = 3;
    const int joffset = 3;
    const int koffset = 1;

    const int width = atoi(argv[1]) + 2 * ioffset;
    const int height = atoi(argv[2]) + 2 * joffset;
    const int depth = atoi(argv[3]) + 2 * koffset;
    const int threads_per_block_x = atoi(argv[4]);
    const int threads_per_block_y = atoi(argv[5]);
    //set threads per block in z direction to zero if axis is set
    const int threads_per_block_z = axis == 0 ? atoi(argv[6]) : 1;
    const int nsteps = atoi(argv[7]);
    const size_t size = width * height * depth;
    const size_t row_byte_size = width * sizeof(REAL_T);
       

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    const int total_threads = threads_per_block_x 
                              * threads_per_block_y 
                              * threads_per_block_z;                    
    if(prop.maxThreadsPerBlock < total_threads) {
        std::cout << "ERROR: max threads per block count("
                  << prop.maxThreadsPerBlock << ") exceeded("
                  << total_threads << ")" << std::endl;
        return -1;
    }

    std::vector< REAL_T > h_data(size, 1);
    std::vector< REAL_T > h_data_in(size, 1);
    std::vector< REAL_T > h_data_out(size, 1);

    // describe data inside texture: 1-component floating point value in this case    
    const int BITS_PER_BYTE = 8;
    hipChannelFormatDesc desc = hipCreateChannelDesc(
                                    sizeof(float) *  BITS_PER_BYTE,
                                    0, 0, 0, hipChannelFormatKindFloat );
    hipArray* d_data_in;
    hipArray* d_data_out;
    CHECK_CUDA(hipMalloc3DArray(&d_data_in, &desc,
                             make_hipExtent(row_byte_size, height, depth),
                             hipArraySurfaceLoadStore));
    CHECK_CUDA(hipMalloc3DArray(&d_data_out, &desc,
                             make_hipExtent(row_byte_size, height, depth),
                             hipArraySurfaceLoadStore));
   
  
    const dim3 threads_per_block = 
        dim3(threads_per_block_x, threads_per_block_y, threads_per_block_z);
    const dim3 offset(ioffset, joffset, koffset);
    const dim3 global_grid_size(width, height, depth);
    
    //compute block size: cover the cases of 2d GPU grid
    //with explicit in-kernel iteration over 3rd dimensions.
    dim3 blocks;
    if(axis == 0 ) {
        //launch on core space only    
        blocks = compute_blocks(dim3(width - 2 * ioffset,
                                     height - 2 * joffset,
                                     depth - 2 * koffset),
                                threads_per_block);
        
    } else if(axis == 'x') {
       //launch on core space only    
       blocks = compute_blocks(dim3(height - 2 * joffset,
                                    depth - 2 * koffset,
                                    1 ),
                               threads_per_block);
    } else if(axis == 'y') {
       //launch on core space only    
       blocks = compute_blocks(dim3(width - 2 * ioffset,
                                    depth - 2 * koffset,
                                    1),
                               threads_per_block);
    } else if(axis == 'z') {
       //launch on core space only    
       blocks = compute_blocks(dim3(width - 2 * ioffset,
                                    height - 2 * joffset,
                                    1),
                                    threads_per_block);
    }
    //from here on all computation is done on the core region only
    //i.e. global grid - halo region

    //fill inner(core) region with zeros
    //note that it is required to re-compute the number of
    //threads per block because the actual threads per block
    //used varies if 2d+in-kernel-iteration is selected
    //just copy from host memory for now 
    //do_all_3d_1_gpu<<<blocks, threads_per_block>>>
    //    (d_data_in, offset, global_grid_size, init<REAL_T>(REAL_T(0)));

    do_all_3d_1_cpu(&h_data_in[0], 
                    offset,
                    global_grid_size,
                    init<REAL_T>(REAL_T(0)));

    hipPitchedPtr host_ptr = 
        make_hipPitchedPtr(&h_data_in[0], row_byte_size, width, height);
    host_ptr.pitch = row_byte_size;   
    hipMemcpy3DParms memcpy_params;
    //configure for device to host copy
    memcpy_params.srcArray = 0;
    memcpy_params.srcPos = make_hipPos(0,0,0);
    memcpy_params.srcPtr = host_ptr;
    memcpy_params.dstArray = d_data_in;
    memcpy_params.dstPos = make_hipPos(0, 0, 0);
    memcpy_params.dstPtr = make_hipPitchedPtr(0, 0, 0, 0);
    memcpy_params.extent = make_hipExtent(width, height, depth);
    memcpy_params.kind = hipMemcpyHostToDevice;


    CHECK_CUDA(hipMemcpy3D(&memcpy_params));   
    memcpy_params.dstArray = d_data_out;
    CHECK_CUDA(hipMemcpy3D(&memcpy_params));    

    //GPU                     
    CUDAEventTimer gpu_timer;
    gpu_timer.start();

    //compute
    if(axis == 0)
        cuda_compute
               (nsteps, d_data_in, d_data_out,
                offset,
                global_grid_size, blocks, threads_per_block,
                diffusion_3d_surface< REAL_T >(),
                do_all_3d_2_gpu_surf< REAL_T, diffusion_3d_surface< REAL_T > >);
    else {
        std::cout << "x, y, z iteration enabled for non-array version only"
                  << std::endl;
    }                                                               
    gpu_timer.stop();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        std::cerr << "ERROR: " << hipGetErrorString(error) << std::endl;
        std::cerr << "Launch config: " 
                  << " blocks: " << blocks << std::endl
                  << " threads per block: " << threads_per_block << std::endl
                  << " grid: " << global_grid_size << std::endl; 
        return -1;
    }


    std::cout << "GPU: " << gpu_timer.elapsed() << std::endl;
    
    //CPU
    Timer cpu_timer;
    cpu_timer.Start();
    REAL_T* out = cpu_compute(nsteps,
                              &h_data_in[0],
                              &h_data_out[0],
                              offset,
                              global_grid_size, diffusion_3d(),
            do_all_3d_2_cpu<REAL_T, diffusion_3d>);
    const double ms = cpu_timer.Stop();
    std::cout << "CPU: " << ms << std::endl;
    
    //copy data back
    host_ptr.ptr = &h_data[0];
    memcpy_params.srcArray = d_data_out;
    memcpy_params.dstArray = 0;
    memcpy_params.dstPtr = host_ptr;
    memcpy_params.srcPtr = make_hipPitchedPtr(0, 0, 0, 0);
    memcpy_params.kind = hipMemcpyDeviceToHost;
    CHECK_CUDA(hipMemcpy3D(&memcpy_params));

    //compare results: h_data holds the data transferred from the GPU
    //                 h_data_out holds the data computed on the CPU  
    std::cout << "Valid: " << std::boolalpha
              << std::equal(h_data.begin(), h_data.end(),
                            out, distance< REAL_T >(EPS))
              << std::endl;
#if 0
    //print something out
               
    do_all_3d_1_cpu(&h_data[0], 
                    offset,
                    global_grid_size,
                    print<REAL_T>());      
    std::cout << "\n=========================================\n";    
    do_all_3d_1_cpu(out, 
                    offset,
                    global_grid_size,
                    print<REAL_T>());      
    std::cout << std::endl;
#endif     
    //free resources
    hipFreeArray(d_data_out);
    hipFreeArray(d_data_in);
    return 0;
}



